#include "hip/hip_runtime.h"
// -*- c++ -*-
// adsyn.cu
// experimental cuda opcodes
//
// V Lazzarini, 2013

#include <csdl.h>
#include <hipfft/hipfft.h>
#define VSAMPS 16
#define MAXBLOCK 8192
#define THREADS_PER_BLOCK 1024

//__shared__ MYFLT mema[64*20];

#define PFRACLO(x)   ((MYFLT)((x) & lomask) * lodiv)

__global__ void component_table(MYFLT *out, int *ndx, MYFLT *tab,
                          float *amp, int *inc, int vsize,
                          int blocks, int lobits, MYFLT lodiv,
                          int lomask) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset, n, lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    n = lndx >> lobits;
    out[i] = amp[h]*(tab[n] +  PFRACLO(lndx)*(tab[n+1] - tab[n]));
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }

}

__global__ void component_sine(MYFLT *out, int *ndx,
                          float *amp, int *inc, int vsize,
                          int blocks) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset, lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    out[i] = amp[h]*sin((PI*2*lndx)/FMAXLEN);
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }
   
}

__global__  void mixdown(MYFLT *out, int comps, int vsize, float kamp){
   int h = threadIdx.x;
   int i;
   for(i=1; i < comps; i++){
     out[h] += out[h + vsize*i];
   }
   out[h] *= kamp;
}


static int destroy_cudaop(CSOUND *csound, void *pp);

typedef struct cudaop_ {
  OPDS h;
  MYFLT *asig;
  MYFLT *kamp, *kfreq, *itabn;
  MYFLT *ftabn, *atabn, *inum;
  MYFLT *out;
  float *amp;
  MYFLT *tab;
  int *ndx, *inc;
  MYFLT *ap, *fp;
  FUNC *itab, *ftab, *atab;
  int N, blocks;
} CUDAOP;

static int init_cudaop(CSOUND *csound, CUDAOP *p){

  int a, b, asize, ipsize, fpsize, tsize;
  int nsmps = CS_KSMPS;
  if(nsmps > 1024) return csound->InitError(csound, "ksmps is too large\n");

  if(*p->itabn != 0){ 
  if((p->itab =
      csound->FTFind(csound, p->itabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->itabn);
  } else p->itab = NULL;

  if((p->ftab =
      csound->FTnp2Find(csound, p->ftabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->ftabn);

  if((p->atab =
      csound->FTnp2Find(csound, p->atabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->atabn);

  a = p->ftab->flen;
  b = p->atab->flen;
  p->N = a < b ? a : b;

  if(*p->inum > 0 && *p->inum < p->N) p->N = *p->inum;

  p->blocks = p->N > THREADS_PER_BLOCK ? p->N/THREADS_PER_BLOCK : 1;

  asize = p->N*nsmps*sizeof(MYFLT);
  ipsize = p->N*sizeof(int);
  fpsize = p->N*sizeof(float);
  if(p->itab)
   tsize = (p->itab->flen+1)*sizeof(MYFLT);

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->amp, fpsize);
  hipMalloc(&p->inc, ipsize);
  if(p->itab) {
   hipMalloc(&p->tab, tsize);
   hipMemcpy(p->tab, p->itab->ftable, tsize, hipMemcpyHostToDevice);
  }
  hipMemset(p->ndx, 0, ipsize);

  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop);
  csound->Message(csound, "%d threads, %d blocks\n", p->N, p->blocks);
  return OK;
}

static void update_params(CSOUND *csound, CUDAOP *p){

  int ipsize = p->N*sizeof(int);
  int fpsize = p->N*sizeof(float);
  float amp[MAXBLOCK];
  int inc[MAXBLOCK], i, j;
  int N = p->N > MAXBLOCK ? MAXBLOCK : p->N;

  for(j=0; N > 0; j++,  N = p->N - N) {
   for(i=0;i < N; i++){
    amp[i] = p->ap[i];
    inc[i] = *p->kfreq*p->fp[i]*FMAXLEN/csound->GetSr(csound);
   }
   hipMemcpy(&p->amp[N*j],amp,fpsize, hipMemcpyHostToDevice);
   hipMemcpy(&p->inc[N*j],inc,ipsize, hipMemcpyHostToDevice);
 }

}

static int perf_cudaop(CSOUND *csound, CUDAOP *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t nsmps = CS_KSMPS;
  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  if (UNLIKELY(offset)) memset(p->asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&(p->asig[nsmps]), '\0', early*sizeof(MYFLT));
  }
 
  update_params(csound, p);
  if(p->itab) 
   component_table<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->tab,p->amp,
                          p->inc,nsmps,
                          p->blocks,
			  p->itab->lobits,
			  p->itab->lodiv,
                          p->itab->lomask);
  else
   component_sine<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->amp,
                          p->inc,nsmps,
                          p->blocks);
   mixdown<<<1,nsmps>>>(p->out,p->N,nsmps,*p->kamp);
   hipMemcpy(p->asig,p->out,nsmps*sizeof(MYFLT),hipMemcpyDeviceToHost);

  return OK;
}

static int destroy_cudaop(CSOUND *csound, void *pp){
  CUDAOP *p = (CUDAOP *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->tab);
  hipFree(p->amp);
  hipFree(p->inc);
  return OK;
}


#include <pstream.h>

typedef struct cudaop2_ {
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  MYFLT *kamp, *kfreq, *itabn;
  MYFLT *inum;
  MYFLT *out;
  float *amp;
  MYFLT *tab;
  int *ndx, *inc;
  float *fp;
  AUXCH out_;
  FUNC *itab;
  int N, blocks;
  int count;
  int vsamps;
  int framecount;
} CUDAOP2;

static int destroy_cudaop2(CSOUND *csound, void *pp);

static int init_cudaop2(CSOUND *csound, CUDAOP2 *p){

  int asize, ipsize, fpsize, tsize;
  if(p->fsig->overlap > 1024)
     return csound->InitError(csound, "overlap is too large\n");

  if((p->itab =
      csound->FTFind(csound, p->itabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->itabn);
  p->N = (p->fsig->N)/2;

  if(*p->inum > 0 && *p->inum < p->N) p->N = *p->inum;

  p->blocks = p->N > THREADS_PER_BLOCK ? p->N/THREADS_PER_BLOCK : 1;
  p->vsamps = p->fsig->overlap < VSAMPS ? VSAMPS : p->fsig->overlap;

  asize = p->N*p->vsamps*sizeof(MYFLT);
  ipsize = p->N*sizeof(int);
  fpsize = p->N*sizeof(float);
  tsize = (p->itab->flen+1)*sizeof(MYFLT);

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->amp, fpsize);
  hipMalloc(&p->inc, ipsize);
  hipMalloc(&p->tab, tsize);
  hipMemset(p->ndx, 0, ipsize);
  hipMemcpy(p->tab,p->itab->ftable,tsize, hipMemcpyHostToDevice);

  asize = p->vsamps*sizeof(MYFLT);
  if(p->out_.auxp == NULL ||
     p->out_.size < asize)
    csound->AuxAlloc(csound, asize , &p->out_);

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop2);
  p->count = 0;

  csound->Message(csound, "%d threads, %d blocks\n", p->N, p->blocks);

  return OK;
}

static void update_params2(CSOUND *csound, CUDAOP2 *p){

  int ipsize = p->N*sizeof(int);
  int fpsize = p->N*sizeof(float);
  float amp[MAXBLOCK];
  int inc[MAXBLOCK], i, j, k;
  int N = p->N > MAXBLOCK ?  MAXBLOCK : p->N;

 for(k=0; N > 0; k++,  N = p->N - N) {
  for(j=i=0;i < N; i++, j+=2){
    amp[i] = p->fp[j];
    inc[i] = MYFLT2LONG(*p->kfreq * p->fp[j+1]*FMAXLEN/csound->GetSr(csound));
  }
  hipMemcpy(&p->amp[k*N],amp,fpsize, hipMemcpyHostToDevice);
  hipMemcpy(&p->inc[k*N],inc,ipsize, hipMemcpyHostToDevice);
 }
}

static int perf_cudaop2(CSOUND *csound, CUDAOP2 *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *out_ = (MYFLT *) p->out_.auxp;
  MYFLT      *asig = p->asig;
  int count = p->count,  vsamps = p->vsamps;
  p->fp = (float *) (p->fsig->frame.auxp);

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n=offset; n < nsmps; n++){
    if(count == 0) {
      update_params2(csound, p);
      component_table<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->tab,p->amp,
                          p->inc,p->vsamps,
                          p->blocks,
                          p->itab->lobits,
                          p->itab->lodiv,
                          p->itab->lomask);
      mixdown<<<1,vsamps>>>(p->out,p->N,vsamps,*p->kamp);
      hipMemcpy(out_,p->out,vsamps*sizeof(MYFLT),hipMemcpyDeviceToHost);
      count = vsamps;
    }
    asig[n] = (MYFLT) out_[vsamps - count];
    count--;
  }
  p->count = count;
  return OK;
}

static int destroy_cudaop2(CSOUND *csound, void *pp){
  CUDAOP2 *p = (CUDAOP2 *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->tab);
  hipFree(p->amp);
  hipFree(p->inc);
  return OK;
}


static OENTRY localops[] = {
  {"cudasynth", sizeof(CUDAOP),0, 5, "a", "kkiiio", (SUBR) init_cudaop, NULL,
   (SUBR) perf_cudaop},
  {"cudasynth", sizeof(CUDAOP2),0, 5, "a", "fkkio", (SUBR) init_cudaop2, NULL,
   (SUBR) perf_cudaop2}
};

extern "C" {
  LINKAGE
}
