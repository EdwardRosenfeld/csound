#include "hip/hip_runtime.h"
// -*- c++ -*-
// adsyn.cu
// experimental cuda opcodes
//
// V Lazzarini, 2013

#include <csdl.h>
#include <pstream.h>

typedef struct cudadsyn_ {
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  MYFLT *kamp, *kfreq;
  MYFLT *inum;
  MYFLT *out;
  float *frame;
  int64_t *ndx;
  float *fp, *previous;
  AUXCH out_;
  int bins, blocks, threads;
  int count;
  int vsamps, mblocks, mthreads;
  int framecount;
} CUDADSYN;

static int destroy_cudadsyn(CSOUND *csound, void *pp);

static int init_cudadsyn(CSOUND *csound, CUDADSYN *p){

  int asize, ipsize, fpsize, blockspt;
  if(p->fsig->overlap > 1024)
     return csound->InitError(csound, "overlap is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 3)
    csound->InitError(csound,
   "this opcode requires device capability 3.0 minimum\n");

  p->bins = (p->fsig->N)/2;

  if(*p->inum > 0 && *p->inum < p->bins) p->bins = *p->inum;

  p->vsamps = p->fsig->overlap;
  p->threads = p->bins*p->vsamps;
  p->blocks = p->threads > blockspt ? p->threads/blockspt : 1;
  p->mthreads = p->bins > p->vsamps ? p->bins : p->vsamps;
  p->mblocks = p->mthreads >  blockspt ? p->mthreads/blockspt : 1;

  p->threads /= p->blocks;
  p->mthreads /= p->mblocks;

  asize = p->bins*p->vsamps*sizeof(MYFLT);
  ipsize  =p->fsig->N*sizeof(int64_t)/2;
  fpsize = p->fsig->N*sizeof(float)*2;

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->frame, fpsize);
  hipMalloc(&p->previous, fpsize);
  hipMemset(p->previous, 0, fpsize);
  hipMemset(p->ndx, 0, ipsize);

  asize = p->vsamps*sizeof(MYFLT);
  if(p->out_.auxp == NULL ||
     p->out_.size < asize)
    csound->AuxAlloc(csound, asize , &p->out_);

  csound->RegisterDeinitCallback(csound, p, destroy_cudadsyn);
  p->count = 0;
  return OK;
}

//__shared__ int64_t ph[2048];

__global__ void sample(MYFLT *out, float *frame, MYFLT pitch, int64_t *ph,
                       float *amps, int bins, int vsize, MYFLT sr) {

  int t = (threadIdx.x + blockIdx.x*blockDim.x);
  int n =  t%vsize;  /* sample index */
  int h = t/vsize;  /* bin index */
  int k = h<<1;
  int64_t lph;
  float a = amps[h], ascl = ((float)n)/vsize;
  MYFLT fscal = pitch*FMAXLEN/sr;
  lph = (ph[h] + (int64_t)(n*round(frame[k+1]*fscal))) & PHMASK;
  a += ascl*(frame[k] - a);
  out[t] = a*sinf((2*PI*lph)/FMAXLEN);
}

__global__ void updatemix(MYFLT *out, float *frame, float *amps, MYFLT kamp,
           int64_t *ph, MYFLT pitch, int bins, int vsize, MYFLT sr){

 int h = threadIdx.x + blockIdx.x*blockDim.x;
 int k = h << 1, i,j;
 /* update phases and amps */
 ph[h]  = (ph[h] + (int64_t)(vsize*round(pitch*frame[k+1]*FMAXLEN/sr))) & PHMASK;
 amps[h] = frame[k];
 if(h >= vsize) 
   return;
 /* mix all partials */
 for(i=1, j= vsize; i < bins; i++, j+=vsize){
    out[h] +=  out[h + j];
  }
 out[h] *= kamp;
}

static int perf_cudadsyn(CSOUND *csound, CUDADSYN *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *out_ = (MYFLT *) p->out_.auxp;
  MYFLT      *asig = p->asig;
  int count = p->count,  vsamps = p->vsamps;
  p->fp = (float *) (p->fsig->frame.auxp);

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
   }

  for(n=offset; n < nsmps; n++){
    if(count == 0) {
      hipMemcpy(p->frame,p->fp,sizeof(float)*p->bins*2,hipMemcpyHostToDevice);
      sample<<<p->blocks,p->threads>>>(p->out,p->frame,
                                               *p->kfreq,
                                                p->ndx,
                                                p->previous,
                                                p->bins,
                                                vsamps,
                                                csound->GetSr(csound));
      if (hipDeviceSynchronize() != hipSuccess)
      csound->Message(csound,"Cuda error: Failed to synchronize\n");
      updatemix<<<p->mblocks,p->mthreads>>>(p->out, p->frame,
                                            p->previous, *p->kamp,
                                            p->ndx,
                                            *p->kfreq,
                                            p->bins,
                                            vsamps,
                                            csound->GetSr(csound));
      hipMemcpy(out_,p->out,vsamps*sizeof(MYFLT),hipMemcpyDeviceToHost);
      count = vsamps;
    }
    asig[n] = (MYFLT) out_[vsamps - count];
    count--;
  }
  p->count = count;
  return OK;
}

static int destroy_cudadsyn(CSOUND *csound, void *pp){
  CUDADSYN *p = (CUDADSYN *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->frame);
  return OK;
}


static OENTRY localops[] = {
  {"cudasynth", sizeof(CUDADSYN),0, 5, "a", "fkko", (SUBR) init_cudadsyn, NULL,
   (SUBR) perf_cudadsyn}
};

extern "C" {
  LINKAGE
}
