#include "hip/hip_runtime.h"
// -*- c++ -*-
// adsyn.cu
// experimental cuda opcodes
//
// V Lazzarini, 2013

#include <csdl.h>
#include <hipfft/hipfft.h>
#define VSAMPS 64

//__shared__ MYFLT mema[64*20];
#define PFRACLO(x)   ((MYFLT)((x) & lomask) * lodiv)

__global__ void component_table(MYFLT *out, int64_t *ndx, MYFLT *tab,
                          float *amp, int *inc, int vsize,
                          int blocks, int lobits, MYFLT lodiv,
                          int lomask) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset, n;
  int64_t lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    n = lndx >> lobits;
    out[i] = amp[h]*(tab[n] +  PFRACLO(lndx)*(tab[n+1] - tab[n]));
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }

}

__global__ void component_sine(MYFLT *out, int64_t *ndx,
                          float *amp, int *inc, int vsize,
                          int blocks) {

  int h = threadIdx.x*blocks + blockIdx.x;
  int i, offset;
  int64_t lndx;
  offset = h*vsize;
  out += offset;

  for(i=0; i < vsize; i++) {
    lndx = ndx[h];
    out[i] = amp[h]*sin((PI*2*lndx)/FMAXLEN);
    ndx[h] = (lndx + inc[h]) & PHMASK;
  }

}

__global__  void mixdown_(MYFLT *out, int comps, int vsize, float kamp){
   int h = threadIdx.x;
   int i;
   for(i=1; i < comps; i++){
     out[h] +=  out[h + vsize*i];
   }
   out[h] *= kamp;
}


static int destroy_cudaop(CSOUND *csound, void *pp);

typedef struct cudaop_ {
  OPDS h;
  MYFLT *asig;
  MYFLT *kamp, *kfreq, *itabn;
  MYFLT *ftabn, *atabn, *inum;
  MYFLT *out;
  float *amp;
  MYFLT *tab;
  int64_t *ndx; 
  int *inc;
  MYFLT *ap, *fp;
  FUNC *itab, *ftab, *atab;
  int N, blocks;
} CUDAOP;

static int init_cudaop(CSOUND *csound, CUDAOP *p){

  int a, b, asize, ipsize, fpsize, tsize;
  int nsmps = CS_KSMPS, blockspt;
  if(nsmps > 1024) return csound->InitError(csound, "ksmps is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 3) 
    csound->InitError(csound, 
     "this opcode requires device capability 3.0 minimum\n");

  if(*p->itabn != 0){
  if((p->itab =
      csound->FTFind(csound, p->itabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->itabn);
  } else p->itab = NULL;

  if((p->ftab =
      csound->FTnp2Find(csound, p->ftabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->ftabn);

  if((p->atab =
      csound->FTnp2Find(csound, p->atabn))== NULL)
    return csound->InitError(csound,
                             "could not find table %.0f\n", *p->atabn);

  a = p->ftab->flen;
  b = p->atab->flen;
  p->N = a < b ? a : b;

  if(*p->inum > 0 && *p->inum < p->N) p->N = *p->inum;

  p->blocks = p->N > blockspt ? p->N/blockspt : 1;

  asize = p->N*nsmps*sizeof(MYFLT);
  ipsize = p->N*sizeof(int64_t);
  fpsize = p->N*sizeof(float);
  if(p->itab)
   tsize = (p->itab->flen+1)*sizeof(MYFLT);

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->amp, fpsize);
  hipMalloc(&p->inc, ipsize);
  if(p->itab) {
   hipMalloc(&p->tab, tsize);
   hipMemcpy(p->tab, p->itab->ftable, tsize, hipMemcpyHostToDevice);
  }
  hipMemset(p->ndx, 0, ipsize);

  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop);
  csound->Message(csound, "%d threads, %d blocks\n", p->N, p->blocks);
  return OK;
}

static void update_params(CSOUND *csound, CUDAOP *p){

  int ipsize = p->N*sizeof(int);
  int fpsize = p->N*sizeof(float);
  float amp[p->N];
  int inc[p->N], i;
  int N = p->N;

   for(i=0;i < N; i++){
    amp[i] = p->ap[i];
    inc[i] = *p->kfreq*p->fp[i]*FMAXLEN/csound->GetSr(csound);
   }
   hipMemcpy(p->amp,amp,fpsize, hipMemcpyHostToDevice);
   hipMemcpy(p->inc,inc,ipsize, hipMemcpyHostToDevice);
 
}

static int perf_cudaop(CSOUND *csound, CUDAOP *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t nsmps = CS_KSMPS;
  p->ap = p->atab->ftable;
  p->fp = p->ftab->ftable;

  if (UNLIKELY(offset)) memset(p->asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&(p->asig[nsmps]), '\0', early*sizeof(MYFLT));
  }

  update_params(csound, p);
  if(p->itab)
   component_table<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->tab,p->amp,
                          p->inc,nsmps,
                          p->blocks,
                          p->itab->lobits,
                          p->itab->lodiv,
                          p->itab->lomask);
  else
   component_sine<<<p->blocks,
        p->N/p->blocks>>>(p->out,p->ndx,
                          p->amp,
                          p->inc,nsmps,
                          p->blocks);
   mixdown_<<<1,nsmps>>>(p->out,p->N,nsmps,*p->kamp);
   hipMemcpy(p->asig,p->out,nsmps*sizeof(MYFLT),hipMemcpyDeviceToHost);

  return OK;
}

static int destroy_cudaop(CSOUND *csound, void *pp){
  CUDAOP *p = (CUDAOP *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->tab);
  hipFree(p->amp);
  hipFree(p->inc);
  return OK;
}


#include <pstream.h>

typedef struct cudaop2_ {
  OPDS h;
  MYFLT *asig;
  PVSDAT *fsig;
  MYFLT *kamp, *kfreq;
  MYFLT *inum;
  MYFLT *out;
  float *frame;
  int64_t *ndx;
  float *fp, *previous;
  AUXCH out_;
  int N, blocks;
  int count;
  int vsamps, vblocks;
  int framecount;
} CUDAOP2;

static int destroy_cudaop2(CSOUND *csound, void *pp);

static int init_cudaop2(CSOUND *csound, CUDAOP2 *p){

  int asize, ipsize, fpsize, blockspt;
  if(p->fsig->overlap > 1024)
     return csound->InitError(csound, "overlap is too large\n");
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  blockspt = deviceProp.maxThreadsPerBlock;
  if(deviceProp.major < 3) 
    csound->InitError(csound, 
   "this opcode requires device capability 3.0 minimum\n");

  p->N = (p->fsig->N)/2;

  if(*p->inum > 0 && *p->inum < p->N) p->N = *p->inum;

  p->blocks = p->N > blockspt ? p->N/blockspt : 1;
  p->vsamps = p->fsig->overlap < VSAMPS ? VSAMPS : p->fsig->overlap;
  p->vblocks = p->vsamps >  blockspt ? p->vsamps/blockspt : 1;

  asize = p->N*p->vsamps*sizeof(MYFLT);
  ipsize = p->N*sizeof(int64_t);
  fpsize = p->N*sizeof(float)*2;

  hipMalloc(&p->out, asize);
  hipMalloc(&p->ndx, ipsize);
  hipMalloc(&p->frame, fpsize);
  hipMalloc(&p->previous, fpsize);
  hipMemset(p->previous, 0, fpsize);
  hipMemset(p->ndx, 0, ipsize);

  asize = p->vsamps*sizeof(MYFLT);
  if(p->out_.auxp == NULL ||
     p->out_.size < asize)
    csound->AuxAlloc(csound, asize , &p->out_);

  csound->RegisterDeinitCallback(csound, p, destroy_cudaop2);
  p->count = 0;
  return OK;
}


__global__ void partial(MYFLT *out, float *frame, MYFLT pitch, int64_t *ph, float *amps,
                      int vsize, MYFLT sr) {

  int h = threadIdx.x + blockIdx.x*blockDim.x;
  int i, offset;
  int64_t inc,lph;
  int k = h << 1;
  float amp = amps[h];
  float ampinc = (frame[k] - amp)/vsize;
  offset = h*vsize;
  out += offset;
  lph = ph[h];
  inc =  round(pitch*frame[k+1]*FMAXLEN/sr);
  for(i=0; i < vsize; i++) {
    out[i] =
    amp*SIN((2*PI*lph)/FMAXLEN);
    lph = (lph + inc) & PHMASK;
    amp += ampinc;
  }
  amps[h] = amp;
  ph[h] = lph;
}

__global__  void mixdown(MYFLT *out, int comps, int vsize, float kamp){
   int h =  threadIdx.x + blockIdx.x*blockDim.x;
   int i;
   for(i=1; i < comps; i++){
     out[h] +=  out[h + vsize*i];
   }
   out[h] *= kamp;
}

__global__ void sample(MYFLT *out, float *frame, MYFLT amp, MYFLT pitch, int64_t *ph, float *amps,
                       int bins, int vsize, MYFLT sr) {

  int n = threadIdx.x + blockIdx.x*blockDim.x, h,k;
  int64_t lph;
  float a, ascl = ((float)n)/vsize;
  MYFLT scal = pitch*FMAXLEN/sr;
  out[n] = 0;
  for(h=k=0; h < bins; h++, k+=2){
          lph = (ph[h] + (int64_t)(n*round(frame[k+1]*scal))) & PHMASK;
          a = amps[h] + ascl*(frame[k] - amps[h]);
          out[n] += a*SIN((2*PI*lph)/FMAXLEN);
  }
  out[n] *= amp;
}

__global__ void update(float *frame, float *amps, int64_t *ph, MYFLT pitch, int vsize, MYFLT sr){

 int h = threadIdx.x + blockIdx.x*blockDim.x;
 int k = h << 1;
 ph[h]  = (ph[h] + (int64_t)(vsize*round(pitch*frame[k+1]*FMAXLEN/sr))) & PHMASK;
 amps[h] =  amps[h] + vsize*(frame[k] - amps[h])/vsize;

}

static int perf_cudaop2(CSOUND *csound, CUDAOP2 *p){

  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  MYFLT *out_ = (MYFLT *) p->out_.auxp;
  MYFLT      *asig = p->asig;
  int count = p->count,  vsamps = p->vsamps;
  p->fp = (float *) (p->fsig->frame.auxp);

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
   }

  for(n=offset; n < nsmps; n++){
    if(count == 0) {
      hipMemcpy(p->frame,p->fp,sizeof(float)*p->N*2,hipMemcpyHostToDevice);
      if(p->N > vsamps) {
       partial<<<p->blocks,p->N/p->blocks>>>
                           (p->out,p->frame,
                            *p->kfreq, p->ndx, p->previous,
                            p->vsamps,
                            csound->GetSr(csound));
       mixdown<<<p->vblocks,vsamps/p->vblocks>>>(p->out,
                                                 p->N,
                                                 vsamps,
                                                 *p->kamp);
      } else {
       sample<<<p->vblocks,vsamps/p->vblocks>>>(p->out,
                                                p->frame,
                                                *p->kamp,
                                                *p->kfreq,
                                                p->ndx,
                                                p->previous,
                                                p->N,
                                                vsamps,
                                                csound->GetSr(csound));
       update<<<p->blocks,p->N/p->blocks>>>(p->frame,
                                            p->previous,
                                            p->ndx,
                                            *p->kfreq,
                                            vsamps,
                                            csound->GetSr(csound));
       }
      hipMemcpy(out_,p->out,vsamps*sizeof(MYFLT),hipMemcpyDeviceToHost);
      count = vsamps;
    }
    asig[n] = (MYFLT) out_[vsamps - count];
    count--;
  }
  p->count = count;
  return OK;
}

static int destroy_cudaop2(CSOUND *csound, void *pp){
  CUDAOP2 *p = (CUDAOP2 *) pp;
  hipFree(p->out);
  hipFree(p->ndx);
  hipFree(p->frame);
  return OK;
}


static OENTRY localops[] = {
  {"cudasynth", sizeof(CUDAOP),0, 5, "a", "kkiiio", (SUBR) init_cudaop, NULL,
   (SUBR) perf_cudaop},
  {"cudasynth", sizeof(CUDAOP2),0, 5, "a", "fkko", (SUBR) init_cudaop2, NULL,
   (SUBR) perf_cudaop2}
};

extern "C" {
  LINKAGE
}
