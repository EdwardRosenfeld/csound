#include "hip/hip_runtime.h"
// -*- c++ -*-
// pconv.cu
// experimental cuda opcodes
//
// V Lazzarini, 2014

#include <csdl.h>
#include <hipfft/hipfft.h>


/* each kernel processes one bin 
*/
__global__ void pconvol(float *out,float *in, 
			float *coef,int rp, int dftsize, 
			int nparts, int end) {
  float re,im,re2,im2;   

  /* thread count */
  int t = (threadIdx.x + blockIdx.x*blockDim.x);

  int k = t<<1;       /* coef pos      */   
  int n = k%dftsize;  /* inframe pos   */

  /* if beyond the buffer end, exit */
  if(k >= end) return;                  
  rp += k/dftsize;       /*  rp pos */

  /* select correct input buffer */
  in += (rp < nparts ? rp : rp%nparts)*dftsize;

  re = coef[k]; im = coef[k+1];
  re2 = in[n];  im2 = in[n+1];
  
  /* complex multiplication + sums
     deal with 0Hz & Nyquist  (n == 0) */
  atomicAdd(&out[n], n  ? re*re2 - im*im2 : re*re2);
  atomicAdd(&out[n+1], n ? re*im2 + re2*im : im*im2);
}  

typedef struct _pconv{
  OPDS h;
  MYFLT *aout, *asig, *ifn, *parts;
  float *out, *coef, *in;
  AUXCH  bufin, bufout;
  int wp, nparts, dftsize, cnt;
  hipfftHandle plan, iplan;
  int threads, blocks;
} PCONV;


int isPowerOfTwo (unsigned int x)
{
  return ((x != 0) && !(x & (x - 1)));
}


static int destroy_pconv(CSOUND *csound, void *pp){
  PCONV *p = (PCONV *) pp;
  hipfftDestroy(p->plan);
  hipfftDestroy(p->iplan);
  hipFree(p->coef);
  hipFree(p->in);
  hipFree(p->out);
  return OK;
}


int pconv_init(CSOUND *csound, PCONV *p){

  FUNC *ftab = csound->FTnp2Find(csound, p->ifn);
  float *tmp;
  int tlen = ftab->flen;
  int end, i, j, k, parts = *p->parts, dftsize, nparts;
  MYFLT *tab = ftab->ftable;

  if(!isPowerOfTwo(parts))
    return csound->InitError(csound, "partition size needs to be power of two\n");

  if(parts > tlen)
     return csound->InitError(csound, "partition size too big \n");

  end = tlen + parts - 1;

  nparts = end / parts;
  dftsize = parts << 1;
  end = nparts*dftsize;

  hipMalloc(&p->coef, sizeof(float)*end);   
  hipMalloc(&p->in, sizeof(float)*end); 
  hipMalloc(&p->out, sizeof(float)*dftsize); 
  hipMemset(p->in,0,sizeof(float)*end);
  hipMemset(p->out, 0, sizeof(float)*dftsize);
  hipMemset(p->coef, 0, sizeof(float)*end);

  p->wp = 0;

  if(!p->bufin.auxp || p->bufin.size < sizeof(float)*dftsize)
     csound->AuxAlloc(csound, sizeof(float)*dftsize, &p->bufin);
  if(!p->bufout.auxp || p->bufout.size < sizeof(float)*dftsize)
     csound->AuxAlloc(csound, sizeof(float)*dftsize, &p->bufout);

  memset(p->bufout.auxp, 0, sizeof(float)*dftsize);

  tmp = (float *) p->bufin.auxp;
  hipfftPlan1d(&p->plan, dftsize, HIPFFT_R2C, 1);
  cufftSetCompatibilityMode(p->plan, CUFFT_COMPATIBILITY_NATIVE);
  hipfftPlan1d(&p->iplan, dftsize, HIPFFT_C2R, 1);
  cufftSetCompatibilityMode(p->iplan, CUFFT_COMPATIBILITY_NATIVE);

  for(i =0, k=0; i < nparts; i++){
    for(j=0; j < dftsize; j++)
      tmp[j] = j < parts && k < tlen ? tab[k++] : 0.f;
    float *pp = p->coef + (nparts - 1 - i)*dftsize;
    hipMemcpy(pp, tmp, sizeof(float)*dftsize, 
               hipMemcpyHostToDevice); 
    hipfftExecR2C(p->plan,pp,(hipfftComplex*)pp);
   }

  hipDeviceSynchronize();
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int blockspt = deviceProp.maxThreadsPerBlock;
 
  end >>= 1;

  p->blocks = end > blockspt ? ceil(end/blockspt) : 1;
  p->threads = end > blockspt ? blockspt : end;

  csound->RegisterDeinitCallback(csound, p, destroy_pconv);

  OPARMS parms;
  csound->GetOParms(csound, &parms);
  if(parms.odebug)
   csound->Message(csound, 
     "blocks %d - threads/block %d - threads %d - dftsize %d\n", 
      p->blocks, p->threads, end, dftsize);

  p->nparts = nparts;
  p->dftsize = dftsize;

  return OK;
}

int pconv_perf(CSOUND *csound, PCONV *p){

  int dftsize = p->dftsize, cnt = p->cnt, wp = p->wp, nparts = p->nparts;
  uint32_t offset = p->h.insdshead->ksmps_offset;
  uint32_t early  = p->h.insdshead->ksmps_no_end;
  uint32_t n, nsmps = CS_KSMPS;
  float *bufin = (float *) p->bufin.auxp, *bufout = (float *) p->bufout.auxp;
  MYFLT *asig = p->asig, *aout = p->aout;
  float *in = p->in, *out = p->out, *coef = p->coef;
  int end = nparts*dftsize;
  int parts = *p->parts;

  if (UNLIKELY(offset)) memset(asig, '\0', offset*sizeof(MYFLT));
  if (UNLIKELY(early)) {
    nsmps -= early;
    memset(&asig[nsmps], '\0', early*sizeof(MYFLT));
  }

  for(n = offset; n < nsmps; n++){
    bufin[cnt] = (float) asig[n];
    aout[n] = (MYFLT) bufout[cnt]/dftsize;
   
    if(++cnt == parts){

       /* in buffer pos */
       int pos = wp*dftsize;

       /* increment delay line pos
          so that it points to the oldest partition
       */
       wp += 1;
       if(wp == nparts) wp = 0;

       /* copy current buffer into newest partition */
       hipMemset(out, 0, sizeof(float)*(dftsize));
       memset(&bufin[parts], 0, sizeof(float)*(parts));
       hipMemcpy(&in[pos],bufin, sizeof(float)*dftsize,hipMemcpyHostToDevice);

       /* apply transform */
       if(hipfftExecR2C(p->plan,&in[pos],(hipfftComplex*)&in[pos])
        != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n");
       if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");

       hipMemset(out, 0, sizeof(float)*dftsize);
       /* convolution */
       pconvol<<<p->blocks,p->threads>>>(out, in, coef, wp, dftsize, nparts, end);
       if (hipDeviceSynchronize() != hipSuccess)
        csound->Message(csound,"Cuda error: Failed to synchronize\n");

       /* transform output */
       if(hipfftExecC2R(p->iplan,(hipfftComplex*)out,out) 
          != HIPFFT_SUCCESS) csound->Message(csound, "cuda fft error\n"); 
 
       /* copy buffer out */
       hipMemcpy(bufin, out, sizeof(float)*dftsize,hipMemcpyDeviceToHost);
 
       /* overlap-save */
       for(int i=0; i < parts; i++) {
	  bufout[i] = bufin[i] + bufout[i+parts];
          bufout[i+parts] = bufin[i+parts];
       }
       cnt = 0;
    }
  }
  p->cnt = cnt;
  p->wp = wp;
  return OK;
}

static OENTRY localops[] = {
  {"cudapconv", sizeof(PCONV),0, 5, "a", "aii", (SUBR) pconv_init, NULL,
    (SUBR) pconv_perf},
};

extern "C" {
  LINKAGE
}
